#include <hip/hip_runtime.h>
#include <errno.h>
#include <stdio.h>
#include <stdlib.h>

int main(int argc, char **argv) {
  float *host, *device;
  hipError_t res;

  long size = sizeof(float) << atol(argv[1]);
  if ((host = (float *)malloc(size)) == NULL) {
    fprintf(stderr, "cpu2gpu fail to allocate host memory\n");
    exit(1);
  }
  fprintf(stderr, "size: %.2fGB\n",
          (double)size / (double)(1 << (10 + 10 + 10)));
  if ((res = hipHostMalloc(&device, size, hipHostMallocDefault)) != hipSuccess) {
    fprintf(stderr, "cpu2gpu hipMalloc failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  if ((res = hipMalloc(&device, size)) != hipSuccess) {
    fprintf(stderr, "cpu2gpu hipMalloc failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  fprintf(stderr, "cpu2gpu start cudaMemcpy\n");
  if ((res = hipMemcpy(device, host, size, hipMemcpyHostToDevice)) !=
      hipSuccess) {
    fprintf(stderr, "cpu2gpu hipMalloc failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  fprintf(stderr, "cpu2gpu end cudaMemcpy\n");
  hipHostFree(host);
  hipFree(device);
}
