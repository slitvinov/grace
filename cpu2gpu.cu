#include <hip/hip_runtime.h>
#include <errno.h>
#include <stdio.h>
#include <stdlib.h>

int main(int argc, char **argv) {
  float *host, *device;
  hipError_t res;

  long size = sizeof(float) << atol(argv[1]);
  if ((host = (float*)malloc(size)) == NULL) {
    fprintf(stderr, "memory: fail to allocate host memory\n");
    exit(1);
  }
  fprintf(stderr, "size: %.2fGB\n",
          (double)size / (double)(1 << (10 + 10 + 10)));
  if ((res = hipHostMalloc(&device, size, hipHostMallocDefault)) != hipSuccess) {
    fprintf(stderr, "memory: hipMalloc failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  if ((res = hipMalloc(&device, size)) != hipSuccess) {
    fprintf(stderr, "memory: hipMalloc failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  fprintf(stderr, "memory: start cudaMemcpy\n");
  if ((res = hipMemcpy(device, host, size, hipMemcpyHostToDevice)) !=
      hipSuccess) {
    fprintf(stderr, "memory: hipMalloc failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  fprintf(stderr, "memory: end cudaMemcpy\n");
  hipHostFree(host);
  hipFree(device);
}
