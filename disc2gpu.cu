#include <hip/hip_runtime.h>
#include <errno.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/mman.h>

int main(int argc, char **argv) {
  float *host, *device;
  int fd;
  unsigned char c = '\0';
  FILE *file;
  hipError_t res;

  long size = sizeof(float) << atol(argv[1]);
  if ((file = fopen("file.raw", "w")) == NULL) {
    fprintf(stderr, "disc2gpu: fail to create file\n");
    exit(1);
  }
  if (fseek(file, size - 1, SEEK_SET) == -1) {
    fprintf(stderr, "disc2gpu: fseek failed\n");
    exit(1);
  }
  if (fwrite(&c, 1, sizeof(c), file) != 1) {
    fprintf(stderr, "disc2gpu: fwrite failed\n");
    exit(1);
  }
  if (fclose(file) != 0) {
    fprintf(stderr, "disc2gpu: fclose failed\n");
    exit(1);
  }
  if ((file = fopen("file.raw", "r+")) == NULL) {
    fprintf(stderr, "disc2gpu: fail reopen\n");
    exit(1);
  }
  if ((fd = fileno(file)) == -1) {
    fprintf(stderr, "disc2gpu: fileno failed\n");
    exit(1);
  }
  host = (float *)mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
  if (host == (void *)-1) {
    fprintf(stderr, "disc2gpu: mmap failed\n");
    fprintf(stderr, "disc2gpu: errno = %d\n", errno);
    exit(1);
  }
  fprintf(stderr, "size: %.2fGB\n",
          (double)size / (double)(1 << (10 + 10 + 10)));
  if ((res = hipMalloc(&device, size)) != hipSuccess) {
    fprintf(stderr, "disc2gpu: hipMalloc failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  fprintf(stderr, "disc2gpu: start cudaMemcpy\n");
  if ((res = hipMemcpy(device, host, size, hipMemcpyHostToDevice)) !=
      hipSuccess) {
    fprintf(stderr, "disc2gpu: hipMalloc failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  fprintf(stderr, "disc2gpu: end cudaMemcpy\n");
  if (fclose(file) != 0) {
    fprintf(stderr, "disc2gpu: fclose failed\n");
    exit(1);
  }
  hipFree(device);
}
