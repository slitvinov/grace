#include <hip/hip_runtime.h>
#include <errno.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/mman.h>

int main(int argc, char **argv) {
  hipError_t res;
  FILE *file;
  float *host, *device;
  int fd;
  long size;

  size = sizeof(float) << atol(argv[1]);
  if ((file = fopen("file.raw", "w")) == NULL) {
    fprintf(stderr, "disc2gpu: error: fopen() filed\n");
    exit(1);
  }
  if (fseek(file, size - 1, SEEK_SET) == -1) {
    fprintf(stderr, "disc2gpu: error:  fseek() failed\n");
    exit(1);
  }
  if (fputc(0, file) == EOF) {
    fprintf(stderr, "disc2gpu: error: fputs() failed\n");
    exit(1);
  }
  if (fclose(file) != 0) {
    fprintf(stderr, "disc2gpu: error: fclose() failed\n");
    exit(1);
  }
  if ((file = fopen("file.raw", "r+")) == NULL) {
    fprintf(stderr, "disc2gpu: error: fail reopen\n");
    exit(1);
  }
  if ((fd = fileno(file)) == -1) {
    fprintf(stderr, "disc2gpu: error: fileno() failed\n");
    exit(1);
  }
  host = (float *)mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
  if (host == (void *)-1) {
    fprintf(stderr, "disc2gpu: error: mmap failed\n");
    fprintf(stderr, "disc2gpu: error: errno = %d\n", errno);
    exit(1);
  }
  memset(host, 0, size);
  fprintf(stderr, "size: %.2fGB\n",
          (double)size / (double)(1 << (10 + 10 + 10)));
  if ((res = hipMalloc(&device, size)) != hipSuccess) {
    fprintf(stderr, "disc2gpu: cudaMalloc() failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  fprintf(stderr, "disc2gpu: start cudaMemcpy\n");
  if ((res = hipMemcpy(device, host, size, hipMemcpyHostToDevice)) !=
      hipSuccess) {
    fprintf(stderr, "disc2gpu: error: cudaMemcpy() failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  fprintf(stderr, "disc2gpu: end cudaMemcpy\n");
  if (fclose(file) != 0) {
    fprintf(stderr, "disc2gpu:  error: fclose() failed\n");
    exit(1);
  }
  hipFree(device);
}
