#include <hip/hip_runtime.h>
#include <errno.h>
#include <stdio.h>
#include <sys/mman.h>

int main(int argc, char **argv) {
  float *host, *device;
  int fd;
  unsigned char c = '\0';
  FILE *file;
  hipError_t res;

  long size = sizeof(float) << atol(argv[1]);
  if ((file = fopen("file.raw", "w")) == NULL) {
    fprintf(stderr, "memory: fail to create file\n");
    exit(1);
  }
  if (fseek(file, size - 1, SEEK_SET) == -1) {
    fprintf(stderr, "memory: fseek failed\n");
    exit(1);
  }
  if (fwrite(&c, 1, sizeof(c), file) != 1) {
    fprintf(stderr, "memory: fwrite failed\n");
    exit(1);
  }
  if (fclose(file) != 0) {
    fprintf(stderr, "memory: fclose failed\n");
    exit(1);
  }
  if ((file = fopen("file.raw", "r+")) == NULL) {
    fprintf(stderr, "memory: fail reopen\n");
    exit(1);
  }
  if ((fd = fileno(file)) == -1) {
    fprintf(stderr, "memory: fileno failed\n");
    exit(1);
  }
  host = (float *)mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
  if (host == (void *)-1) {
    fprintf(stderr, "memory: mmap failed\n");
    fprintf(stderr, "memory: errno = %d\n", errno);
    exit(1);
  }
  fprintf(stderr, "size: %.2fGB\n",
          (double)size / (double)(1 << (10 + 10 + 10)));
  if ((res = hipMalloc(&device, size)) != hipSuccess) {
    fprintf(stderr, "memory: hipMalloc failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  if ((res = hipMemcpy(device, host, size, hipMemcpyHostToDevice)) !=
      hipSuccess) {
    fprintf(stderr, "memory: hipMalloc failed: '%s'\n",
            hipGetErrorString(res));
    exit(1);
  }
  if (fclose(file) != 0) {
    fprintf(stderr, "memory: fclose failed\n");
    exit(1);
  }
  hipFree(device);
}
